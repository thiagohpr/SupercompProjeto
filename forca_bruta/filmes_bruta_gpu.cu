#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <fstream>


#define RESET   "\033[0m"
#define BLACK   "\033[30m"      /* Black */
#define RED     "\033[31m"      /* Red */
#define GREEN   "\033[32m"      /* Green */
#define YELLOW  "\033[33m"      /* Yellow */
#define BLUE    "\033[34m"      /* Blue */
#define MAGENTA "\033[35m"      /* Magenta */
#define CYAN    "\033[36m"      /* Cyan */
#define WHITE   "\033[37m"      /* White */
#define BOLDBLACK   "\033[1m\033[30m"      /* Bold Black */
#define BOLDRED     "\033[1m\033[31m"      /* Bold Red */
#define BOLDGREEN   "\033[1m\033[32m"      /* Bold Green */
#define BOLDYELLOW  "\033[1m\033[33m"      /* Bold Yellow */
#define BOLDBLUE    "\033[1m\033[34m"      /* Bold Blue */
#define BOLDMAGENTA "\033[1m\033[35m"      /* Bold Magenta */
#define BOLDCYAN    "\033[1m\033[36m"      /* Bold Cyan */
#define BOLDWHITE   "\033[1m\033[37m"      /* Bold White */

struct filme {
    int id;
    int h_inicio;
    int h_fim;
    int categoria;
    int duracao;
};

using namespace std;

void generateOuput(vector<filme> mochila, int n_filme, int n_cat){
    ofstream outputFile;
    outputFile.open("../outputs2/output_exau_"+to_string(n_filme)+"_"+to_string(n_cat));
    outputFile << n_filme << " " << n_cat << endl;

    double tempo_tela=0;

    int n_mochila = 0;
    for(auto& this_film : mochila){
        tempo_tela +=this_film.duracao;
        n_mochila += 1;
    }
    double media = tempo_tela/(double)n_mochila;
    outputFile << n_mochila << " " << tempo_tela << " " << media << endl;
    outputFile.close();

}

void output_visual(vector<filme> mochila){
    cout<<"\n\n\n";
    
    cout << "|-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------|" << endl;
    cout << "|00\t01\t02\t03\t04\t05\t06\t07\t08\t09\t10\t11\t12\t13\t14\t15\t16\t17\t18\t19\t20\t21\t22\t23\t|" << endl;
    int agora = 0;
    int numero_de_espacos = 0;
    int numero_de_filme = 0;
    for(auto& this_film : mochila){

        agora = this_film.h_inicio;
        numero_de_espacos = agora;
        numero_de_filme = this_film.h_fim - this_film.h_inicio;
        for(int i = 0; i<numero_de_espacos; i++){
            cout<<"\t";
        }

        for(int i = 0; i<numero_de_filme; i++){
            if(this_film.categoria==1){
                cout << RED;
            }
            else if (this_film.categoria==2){
                cout << GREEN;
            }
            else if (this_film.categoria==3){
                cout << MAGENTA;
            }
            else if (this_film.categoria==4){
                cout << YELLOW;
            }
            else if (this_film.categoria==5){
                cout << BLUE;
            }
            else if (this_film.categoria==6){
                cout << MAGENTA;
            }
            else if (this_film.categoria==7){
                cout << CYAN;
            }
            
            cout<<"|"<<"|"<<"|"<<"|"<<"|"<<"|"<<"|"<<"|";
            cout << RESET;
        }
        cout<<endl;
        
    }
    cout << "|-----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------|" << endl;

    // for(auto& this_film : mochila){
    //     cout << this_film.h_inicio << " " << this_film.h_fim << " " << this_film.id << '\n';
    // }
}

void get_combinations(vector<filme> movies, vector<vector<filme>> &combinations)
{
  int slent = pow(2, movies.size());

  for (int i = 0; i < slent; i++)
  {
    vector<filme> temp;
    for (int j = 0; j < movies.size(); j++)
    {
      if ((i & int(pow(2, j))))
      {
        temp.push_back(movies[j]);
        if (temp.size() > 25) break;
      }
    }
    if (temp.size() > 0)
    {
      combinations.push_back(temp);
    }
  }

  sort(combinations.begin(), combinations.end(), [](auto& i, auto& j){return i.size() > j.size();});

  return;             
}


// Functor for verifying agenda constraints
struct VerifyAgenda {
    const thrust::device_vector<bool>& agenda;

    __host__ __device__
    bool operator()(const filme& this_filme) const {
        if (this_filme.h_inicio == this_filme.h_fim)
            return !agenda[this_filme.h_inicio];

        for (int i = this_filme.h_inicio; i < this_filme.h_fim; i++) {
            if (agenda[i] == true) {
                return false;
            }
        }
        return true;
    }
};

// Functor for filling the agenda
struct FillAgenda {
    thrust::device_vector<bool>& agenda;

    __host__ __device__
    void operator()(const filme& this_filme) const {
        if (this_filme.h_inicio == this_filme.h_fim)
            agenda[this_filme.h_inicio] = true;
        else {
            for (int i = this_filme.h_inicio; i < this_filme.h_fim; i++) {
                agenda[i] = true;
            }
        }
    }
};

// Functor for calculating the duration of a film
struct CalculateDuration {
    __host__ __device__
    int operator()(const filme& this_filme) const {
        return this_filme.duracao;
    }
};

int main() {
    int n_filme = 0;
    int n_cat = 0;
    vector<filme> todos_filmes;

    vector<int> max_categorias;

    cin >> n_filme >> n_cat;

    todos_filmes.reserve(n_filme);
    int this_max_cat = 0;
    for (int i = 0; i < n_cat; i++) {
        cin >> this_max_cat;
        max_categorias.push_back(this_max_cat);
    }

    int this_h_inicio = 0;
    int this_h_fim;
    int this_categoria = 0;

    for (int i = 0; i < n_filme; i++) {
        cin >> this_h_inicio >> this_h_fim >> this_categoria;
        if (this_h_fim < this_h_inicio) {
            this_h_fim += 24;
        }
        todos_filmes.push_back({ i, this_h_inicio, this_h_fim, this_categoria, this_h_fim - this_h_inicio });
    }

    vector<vector<filme>> todas_possibilidades;
    get_combinations(todos_filmes, todas_possibilidades);

    double melhor_tempo_tela = 0;
    vector<filme> melhor_solucao;

    thrust::host_vector<bool> melhor_agenda_host(24, false);
    thrust::device_vector<bool> melhor_agenda = melhor_agenda_host;

    for (auto& solucao : todas_possibilidades) {
        thrust::device_vector<bool> agenda(24, false);
        thrust::device_vector<int> max_categorias_copia = max_categorias;
        thrust::device_vector<filme> d_solucao = solucao;

        // Verify agenda and subtract category counts
        auto valid_solution_end = thrust::remove_if(d_solucao.begin(), d_solucao.end(), VerifyAgenda{ agenda });
        valid_solution_end = thrust::remove_if(d_solucao.begin(), valid_solution_end, [&max_categorias_copia] __device__(const filme& this_filme) {
            return max_categorias_copia[this_filme.categoria - 1]-- <= 0;
        });
        d_solucao.erase(valid_solution_end, d_solucao.end());

        // Fill agenda for valid films
        thrust::for_each(d_solucao.begin(), d_solucao.end(), FillAgenda{ agenda });

        // Calculate total duration
        double tempo_tela = thrust::transform_reduce(d_solucao.begin(), d_solucao.end(), CalculateDuration(), 0, thrust::plus<int>());

        if (tempo_tela > melhor_tempo_tela) {
            melhor_tempo_tela = tempo_tela;
            melhor_solucao = solucao;
            melhor_agenda = agenda;
        }
    }

    // generateOuput(melhor_solucao, n_filme, n_cat);

    cout << '\n';
    return 0;
}
