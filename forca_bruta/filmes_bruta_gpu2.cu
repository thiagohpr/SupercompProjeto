#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <map>
#include <fstream>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>

using namespace std;

struct filme {
    int id;
    int h_inicio;
    int h_fim;
    int categoria;
    int duracao;
};

void generateOuput(int n_adicionados, int n_filme, int n_cat) {
    ofstream outputFile;
    outputFile.open("./outputs/output_gpu_" + to_string(n_filme) + "_" + to_string(n_cat));
    outputFile << n_filme << " " << n_cat << endl;

    outputFile << n_adicionados << endl;
    outputFile.close();
}

struct get_combinations_max_movies 
{
   int *max_categorias;
   filme *todos_filmes;
   int n_filmes;
   int n_cat;


    get_combinations_max_movies(int _n_filmes, filme *_todos_filmes, int _n_cat ,int *_max_categorias) : n_filmes(_n_filmes), todos_filmes(_todos_filmes), n_cat(_n_cat) ,max_categorias(_max_categorias) {};
   __device__ __host__
   int operator() (int slent)
   {  

      bool agenda[30];
      for(int h = 0; h < 30; h++) {
        agenda[h] = false;
      }

      int max_categorias_cp[30];
      for(int h = 0; h <= n_cat; h++) {
        max_categorias_cp[h] = max_categorias[h];
      }

      int n_filmes_assistidos = 0;
      for(int i = 0; i < n_filmes; i++)
      {
          if ((slent & int(pow(2, i))))
          {
              // Verificação da agenda
              if(todos_filmes[i].h_inicio == todos_filmes[i].h_fim) {
                if(agenda[todos_filmes[i].h_inicio]) return -1;
                agenda[todos_filmes[i].h_inicio] = true;
              } 
              else {
                for(int j = todos_filmes[i].h_inicio; j < todos_filmes[i].h_fim; j++) {
                  if(agenda[j]) return -1;
                  agenda[j] = true;
                }
              }
              if(max_categorias_cp[todos_filmes[i].categoria] == 0) return -1;
              max_categorias_cp[todos_filmes[i].categoria] --;
              n_filmes_assistidos++;
          }
      }
      return n_filmes_assistidos;
   }
};

int main() {
    int n_filme = 0;
    int n_cat = 0;
    vector<filme> todos_filmes;

    vector<int> max_categorias;

    cin >> n_filme >> n_cat;

    todos_filmes.reserve(n_filme);
    int this_max_cat = 0;
    for (int i = 0; i < n_cat; i++) {
        cin >> this_max_cat;
        max_categorias.push_back(this_max_cat);
    }

    int this_h_inicio = 0;
    int this_h_fim;
    int this_categoria = 0;

    for (int i = 0; i < n_filme; i++) {
        cin >> this_h_inicio >> this_h_fim >> this_categoria;
        if (this_h_fim < this_h_inicio) {
            this_h_fim += 24;
        }
        todos_filmes.push_back({ i, this_h_inicio, this_h_fim, this_categoria, this_h_fim - this_h_inicio });
    }

    thrust::device_vector<int> cats_lim_gpu(max_categorias);

    thrust::device_vector<filme> mov_gpu(todos_filmes);

    thrust::device_vector<int> mov_count(pow(2, todos_filmes.size()));
    
    thrust::counting_iterator<int> comb(0);

    thrust::transform(comb, comb + pow(2, todos_filmes.size()), mov_count.begin(), get_combinations_max_movies(todos_filmes.size(), thrust::raw_pointer_cast(mov_gpu.data()), n_cat ,thrust::raw_pointer_cast(cats_lim_gpu.data())));

    int best = *thrust::max_element(mov_count.begin(), mov_count.end());

    generateOuput(best, n_filme, n_cat);

    cout << "best: " <<  best << endl;
    return 0;
}

