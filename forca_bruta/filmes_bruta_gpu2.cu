#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <map>
#include <fstream>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>

using namespace std;

struct filme {
    int id;
    int h_inicio;
    int h_fim;
    int categoria;
    int duracao;
};

void generateOuput(vector<filme> mochila, int n_filme, int n_cat) {
    ofstream outputFile;
    outputFile.open("../outputs2/output_exau_" + to_string(n_filme) + "_" + to_string(n_cat));
    outputFile << n_filme << " " << n_cat << endl;

    double tempo_tela = 0;

    int n_mochila = 0;
    for (auto& this_film : mochila) {
        tempo_tela += this_film.duracao;
        n_mochila += 1;
    }
    double media = tempo_tela / (double)n_mochila;
    outputFile << n_mochila << " " << tempo_tela << " " << media << endl;
    outputFile.close();
}

struct verify_agenda_functor {
    const thrust::device_vector<bool>& agenda;
    verify_agenda_functor(const thrust::device_vector<bool>& _agenda) : agenda(_agenda) {}

    __host__ __device__
    bool operator()(const filme& this_filme) {
        if (this_filme.h_inicio == this_filme.h_fim)
            return !agenda[this_filme.h_inicio];

        for (int i = this_filme.h_inicio; i < this_filme.h_fim; i++) {
            if (agenda[i] == true) {
                return false;
            }
        }
        return true;
    }
};

struct fill_agenda_functor {
    thrust::device_vector<bool>& agenda;
    fill_agenda_functor(thrust::device_vector<bool>& _agenda) : agenda(_agenda) {}

    __host__ __device__
    void operator()(const filme& this_filme) {
        if (this_filme.h_inicio == this_filme.h_fim)
            agenda[this_filme.h_inicio] = true;
        else {
            for (int i = this_filme.h_inicio; i < this_filme.h_fim; i++) {
                agenda[i] = true;
            }
        }
    }
};

struct CalculateDuration {
    __host__ __device__
    int operator()(const filme& this_filme) const {
        return this_filme.duracao;
    }
};

void get_combinations(const vector<filme>& movies, vector<vector<filme>>& combinations) {
    int slent = pow(2, movies.size());

    for (int i = 0; i < slent; i++) {
        vector<filme> temp;
        for (int j = 0; j < movies.size(); j++) {
            if ((i & int(pow(2, j)))) {
                temp.push_back(movies[j]);
                if (temp.size() > 25) break;
            }
        }
        if (temp.size() > 0) {
            combinations.push_back(temp);
        }
    }

    sort(combinations.begin(), combinations.end(), [](const auto& i, const auto& j) { return i.size() > j.size(); });
}

int main() {
    int n_filme = 0;
    int n_cat = 0;
    vector<filme> todos_filmes;

    vector<int> max_categorias;

    cin >> n_filme >> n_cat;

    todos_filmes.reserve(n_filme);
    int this_max_cat = 0;
    for (int i = 0; i < n_cat; i++) {
        cin >> this_max_cat;
        max_categorias.push_back(this_max_cat);
    }

    int this_h_inicio = 0;
    int this_h_fim;
    int this_categoria = 0;

    for (int i = 0; i < n_filme; i++) {
        cin >> this_h_inicio >> this_h_fim >> this_categoria;
        if (this_h_fim < this_h_inicio) {
            this_h_fim += 24;
        }
        todos_filmes.push_back({ i, this_h_inicio, this_h_fim, this_categoria, this_h_fim - this_h_inicio });
    }

    vector<vector<filme>> todas_possibilidades;
    get_combinations(todos_filmes, todas_possibilidades);

    double melhor_tempo_tela = 0;
    vector<filme> melhor_solucao;

    for (auto& solucao : todas_possibilidades) {
        thrust::device_vector<filme> d_solucao(solucao.begin(), solucao.end());
        thrust::device_vector<bool> agenda(24, false);
        thrust::device_vector<int> max_categorias_copia(max_categorias.begin(), max_categorias.end());

        bool valid_solution = true;

        verify_agenda_functor verify_functor(agenda);
        fill_agenda_functor fill_functor(agenda);

        auto valid_solution_end = thrust::remove_if(d_solucao.begin(), d_solucao.end(), verify_functor);

        if (valid_solution_end != d_solucao.end()) {
            valid_solution = false;
        }

        if (valid_solution) {
            double tempo_tela = thrust::transform_reduce(d_solucao.begin(), d_solucao.end(),CalculateDuration(), 0.0, thrust::plus<double>());

            if (tempo_tela > melhor_tempo_tela) {
                melhor_tempo_tela = tempo_tela;
                thrust::copy(d_solucao.begin(), d_solucao.end(), melhor_solucao.begin());
            }
        }
    }

    generateOuput(melhor_solucao, n_filme, n_cat);

    cout << '\n';
    return 0;
}
